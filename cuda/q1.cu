
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
                    double num1;
                    double num2;
                    double result;
};



__device__ void function(struct num_array *a)
{
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    return;
}
__global__ void calculate(char *mem, int num)
{
      int bs = blockDim.x * blockDim.y;
      int i = bs * blockIdx.x + blockDim.x * threadIdx.y + threadIdx.x;
      if(i >= num)
           return;
       struct num_array *a = (struct num_array *)(mem + (i * 3 * sizeof(double)));
      function(a);
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i,trow,tcol,block_size;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks;

    if(argc == 4){
         num = atoi(argv[1]);   /*Update after checking*/
         if(num <= 0)
               num = NUM;
	 trow = atoi(argv[2]);
	 tcol = atoi(argv[3]);
	 block_size = trow*tcol;
	 if( block_size > 1024 || block_size <= 0){
	    printf("Row and column size should be between 1 and 32");
	    return 1; 
	}
    }
     else{
	printf("Correct Usage - ./q1 {number of elements} {rows} {cols}");
	return 2;
    }
    /* Allocate host (CPU) memory and initialize*/

    ptr = (char *)malloc(num * 3 * sizeof(double));
    sptr = ptr; 
    for(i=0; i<num; ++i){
       pa = (struct num_array *) sptr;
       pa->num1 = (double) i + (double) i * 0.1;
       pa->num2 = pa->num1 + 1.0;
       sptr += 3 * sizeof(double);
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = num /block_size;
    
    if(num % block_size)
           ++blocks;

    calculate<<<blocks, dim3(tcol,trow,1)>>>(gpu_mem, num);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;
   
    /*Print the last element for sanity check*/ 
    pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("num1=%f num2=%f result=%f\n", pa->num1, pa->num2, pa->result);

    
    free(ptr);
}
