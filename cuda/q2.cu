
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

/*struct num_array{
                    double num1;
                    double num2;
                    double result;
};*/



/*__device__ void xor(int *x, int *y)
{
    	int a = *x & *y;
	int b = ~(*x) & ~(*y);
	int r = ~a & ~b;
	*x = r;
	return;
}*/


__global__ void calculate(int *mem, int num, int o)
{
      
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      if(i > num)
           return;
      // struct num_array *a = (struct num_array *)(mem + (i * 3 * sizeof(double)));
      
	int a = *(mem + i) & *( mem + i + num);
	int b = ~*(mem + i) & ~*(mem + i + num);
	*(mem + i) = ~a & ~b;

	if( blockIdx.x == 0 && threadIdx.x == 0 && o == 1){
	//	printf( "%d \n",*(mem + 2*num ));
		*(mem + num ) = *(mem + 2*num );
	//	printf( "%d \n",*(mem + num ));

	}
      
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i,seed;
    //struct num_array *pa;
    int *ptr;
    int *sptr;
    int *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks;

    if(argc == 3){
         num = atoi(argv[1]);   /*Update after checking*/
         if(num <= 0)
               num = NUM;
	 seed = atoi(argv[2]);
    }
     else{
	printf("Correct Usage - ./q2 {number of elements} {seed} \n");
	return 1;
    }
    /* Allocate host (CPU) memory and initialize*/
    srand(seed);
    ptr = (int *)malloc(num * sizeof(int));
    sptr = ptr; 
    for(i=0; i<num; ++i){
       //pa = (struct num_array *) sptr;
       //pa->num1 = (double) i + (double) i * 0.1;
       //pa->num2 = pa->num1 + 1.0;
       *sptr = rand();
	//if( i == num - 1)
	//	printf("last no is %d \n", *sptr);
       sptr += 1;
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    int s_n = num/2;
    int o = 0;
    if(num % 2)
	o = 1;
    i = s_n;
    while( i>=1)
    {
   
	blocks = i /1024;
    
    	if(i % 1024)
           ++blocks;
	//printf("The call is %d %d \n",i,o);
    	calculate<<<blocks, dim3(1024,1,1)>>>(gpu_mem, i,o);
    	CUDA_ERROR_EXIT("kernel invocation");
    	gettimeofday(&end, NULL);
	//cudaDeviceSynchronize();
	i = i + o;

    	if ( i % 2)
		o = 1;
    	else
		o = 0;
	i = i/2;
 
    }
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    //printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    //sptr = ptr;
   
    /*Print the result*/ 
    //pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("xor sum = %d\n", *(ptr));

    
    free(ptr);
}
